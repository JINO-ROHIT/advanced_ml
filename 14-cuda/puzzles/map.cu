// Implement a "kernel" (GPU function) that adds 10 to each position of vector a and stores it in vector out. You have 1 thread per position.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int DSIZE = 1024;

__global__ void map(float *a, int ds){
    int idx = threadIdx.x;
    if (idx < ds){
        a[idx] += 10;
    }
}

int main(){
    float *h_A;
    float *d_A;

    h_A = new float[DSIZE];

    for (int i = 0; i < DSIZE; i++)
    {
        h_A[i] = rand() / (float) RAND_MAX;
    }


    hipMalloc(&d_A, DSIZE * sizeof(float));

    hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);

    map<<<1, DSIZE>>>(d_A, DSIZE);

    hipDeviceSynchronize();

    hipMemcpy(h_A, d_A, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("h_A[%d] = %f\n", i, h_A[i]);
    }

    hipFree(d_A);

    delete[] h_A;
}
