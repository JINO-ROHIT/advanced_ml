// Implement a kernel that adds 10 to each position of a and stores it in out. You have more threads than positions. (dont use a block, since use many threads)

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int DSIZE = 1000; 
const int THREADS_COUNT = 1024;

__global__ void add_ten(const float *a, float *out, int ds){
    int idx = threadIdx.x;

    if (idx < ds) {
        out[idx] = a[idx] + 10.0f;
    }
}

int main(){
    float *h_A, *h_out;
    float *d_A, *d_out;

    h_A = new float[DSIZE];
    h_out = new float[DSIZE];

    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand() / (float) RAND_MAX;
    }

    hipMalloc(&d_A, DSIZE * sizeof(float));
    hipMalloc(&d_out, DSIZE * sizeof(float));

    hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);

    add_ten<<<1, THREADS_COUNT>>>(d_A, d_out, DSIZE);

    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("h_out[%d] = %f\n", i, h_out[i]);
    }

    hipFree(d_A);
    hipFree(d_out);
    delete[] h_A;
    delete[] h_out;

    return 0;
}
