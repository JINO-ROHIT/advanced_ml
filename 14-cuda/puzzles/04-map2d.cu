#include "hip/hip_runtime.h"
// Implement a kernel that adds 10 to each position of a and stores it in out. Input a is 2D and square. You have more threads than positions.(again dont use a block)

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int M = 1000;
const int N = 1000;
const int THREADS_PER_DIM = 32;

__global__ void add_ten(const float *a, float *out, int row, int col) {
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;

    int curr_idx = idx_x * row + idx_y;

    if (idx_x < row && idx_y < col) {
        out[curr_idx] = a[curr_idx] + 10;
    }
}

int main() {
    float *h_A, *h_out;
    float *d_A, *d_out;
    
    h_A = new float[M * N];
    h_out = new float[M * N];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            h_A[i * N + j] = rand() / (float)RAND_MAX;
        }
    }
    
    hipMalloc(&d_A, M * N * sizeof(float));
    hipMalloc(&d_out, M * N * sizeof(float));
    
    hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threads(THREADS_PER_DIM, THREADS_PER_DIM);
    add_ten<<<1, threads>>>(d_A, d_out, M, N);
    
    hipDeviceSynchronize();
    
    hipMemcpy(h_out, d_out, M * N * sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < min(5, M); i++) {
        for (int j = 0; j < min(5, N); j++) {
            printf("h_out[%d][%d] = %f\n", i, j, h_out[i * N + j]);
        }
    }
    
    hipFree(d_A);
    hipFree(d_out);
    delete[] h_A;
    delete[] h_out;
}