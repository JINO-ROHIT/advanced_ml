// Implement a kernel that sums together the last 3 position of a and stores it in out. You have 1 thread per position.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int DSIZE = 1000; 
const int THREADS_PER_BLOCK = 256; 

__global__ void pooling(const float *a, float *out, int ds) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx <= ds - 3) { 
        out[idx] = a[idx] + a[idx + 1] + a[idx + 2];
    } else if (idx < ds) {
        out[idx] = 0;
    }
}

int main(){
    float *h_A, *h_out;
    float *d_A, *d_out;


    h_A = new float[DSIZE];
    h_out = new float[DSIZE];


    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand() / (float) RAND_MAX; 
    }


    hipMalloc(&d_A, DSIZE * sizeof(float));
    hipMalloc(&d_out, DSIZE * sizeof(float));

    hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);


    int numBlocks = (DSIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    pooling<<<numBlocks, THREADS_PER_BLOCK>>>(d_A, d_out, DSIZE);

    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("h_out[%d] = %f\n", i, h_out[i]);
    }

    hipFree(d_A);
    hipFree(d_out);
    delete[] h_A;
}