// Implement a kernel that adds together each position of a and b and stores it in out. You have 1 thread per position.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int DSIZE = 1024;

__global__ void zip(float *A, float *B, float *C, int ds) {
    int idx = threadIdx.x;
    if (idx < ds) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    h_A = new float[DSIZE];
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];

    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    hipMalloc(&d_A, sizeof(float) * DSIZE);
    hipMalloc(&d_B, sizeof(float) * DSIZE);
    hipMalloc(&d_C, sizeof(float) * DSIZE);

    hipMemcpy(d_A, h_A, sizeof(float) * DSIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * DSIZE, hipMemcpyHostToDevice);

    zip<<<1, DSIZE>>>(d_A, d_B, d_C, DSIZE);

    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, sizeof(float) * DSIZE, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("C[%d] = %f \n", i, h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
}
