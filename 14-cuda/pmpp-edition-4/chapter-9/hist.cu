#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define BIN_SIZE 4
#define NUM_BINS ((26 + BIN_SIZE - 1) / BIN_SIZE)
#define CFACTOR 32
#define BLOCKS_PER_SM 32

#define CUDA_CHECK(call)                                                                                 \
    do {                                                                                                 \
        hipError_t error = call;                                                                        \
        if (error != hipSuccess) {                                                                      \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE);                                                                          \
        }                                                                                                \
    } while (0)

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

int getNumSMs() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Get properties of device 0
    return prop.multiProcessorCount;
}

void clear_l2() {
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 2;
        gpuErrchk(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    gpuErrchk(hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size));
}

char* generate_random_text(unsigned int length) {
    char* text = (char*)malloc((length + 1) * sizeof(char));
    if (text == NULL) {
        printf("Memory allocation failed!\n");
        return NULL;
    }

    srand((unsigned int)time(NULL));
    for (unsigned int i = 0; i < length; i++) {
        text[i] = 'a' + (rand() % 26);
    }
    text[length] = '\0';
    return text;
}

void print_histogram_bins(const unsigned int* histo, const char* label) {
    printf("\n%s:\n", label);
    for (int i = 0; i < NUM_BINS; i++) {
        printf("Bin %d (letters %c-%c): %u\n", i, 'a' + (i * BIN_SIZE), 'a' + min(25, (i + 1) * BIN_SIZE - 1),
               histo[i]);
    }
}

// kernels
// -------------------------------------------------------------------------------------------------------------------

__global__ void histo_kernel(char* data, unsigned int length, unsigned int* histo) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&histo[alphabet_position / BIN_SIZE], 1);
        }
    }
}

__global__ void histo_private_kernel(char* data, unsigned int length, unsigned int* histo) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&histo[blockIdx.x * NUM_BINS + alphabet_position / BIN_SIZE], 1);
        }
    }

    if (blockIdx.x > 0) {
        __syncthreads();
        for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
            unsigned int binValue = histo[blockIdx.x * NUM_BINS + bin];
            if (binValue > 0) {
                atomicAdd(&histo[bin], binValue);
            }
        }
    }
}

__global__ void histo_private_kernel_shared_memory(char* data, unsigned int length, unsigned int* histo) {
    // Initialize privatized bins in the shared memory
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&histo_s[alphabet_position / BIN_SIZE], 1);
        }
    }
    __syncthreads();

    // Commit to the global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        unsigned int binValue = histo_s[bin];
        if (binValue > 0) {
            atomicAdd(&histo[bin], binValue);
        }
    }
}

__global__ void histo_private_kernel_thread_coarsing(char* data, unsigned int length, unsigned int* histo) {
    // Initialize privatized bins in the shared memory
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int i = tid * CFACTOR; i < min((tid + 1) * CFACTOR, length); ++i) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&histo_s[alphabet_position / BIN_SIZE], 1);
        }
    }
    __syncthreads();

    // Commit to the global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        unsigned int binValue = histo_s[bin];
        if (binValue > 0) {
            atomicAdd(&histo[bin], binValue);
        }
    }
}

__global__ void histo_private_kernel_thread_coarsing_and_coalesced_memory_accesss(char* data, unsigned int length,
                                                                                  unsigned int* histo) {
    // Initialize privatized bins in the shared memory
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // coalesced memory access pattern
    for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&histo_s[alphabet_position / BIN_SIZE], 1);
        }
    }
    __syncthreads();

    // Commit to the global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        unsigned int binValue = histo_s[bin];
        if (binValue > 0) {
            atomicAdd(&histo[bin], binValue);
        }
    }
}



// kernel functions
// -------------------------------------------------------------------------------------------------------------------


inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}

void histogram_parallel(char* data, unsigned int length, unsigned int* histo) {
    char* d_data;
    unsigned int* d_histo;

    CUDA_CHECK(hipMalloc((void**)&d_data, length * sizeof(char)));
    CUDA_CHECK(hipMemcpy(d_data, data, length * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&d_histo, NUM_BINS * sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_histo, 0, NUM_BINS * sizeof(unsigned int)));

    dim3 dimBlock(1024);
    dim3 dimGrid(cdiv(length, dimBlock.x));

    histo_kernel<<<dimGrid, dimBlock>>>(d_data, length, d_histo);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(histo, d_histo, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_histo));
}

void histogram_parallel_private(char* data, unsigned int length, unsigned int* histo) {
    char* d_data;
    unsigned int* d_histo;

    dim3 dimBlock(1024);
    dim3 dimGrid(cdiv(length, dimBlock.x));

    CUDA_CHECK(hipMalloc((void**)&d_data, length * sizeof(char)));
    CUDA_CHECK(hipMemcpy(d_data, data, length * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&d_histo,
                          NUM_BINS * dimGrid.x * sizeof(unsigned int)));  // here we allocate NUM_BINS for every block

    histo_private_kernel<<<dimGrid, dimBlock>>>(d_data, length, d_histo);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(histo, d_histo, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_histo));
}

void histogram_parallel_private_with_shared_memory(char* data, unsigned int length, unsigned int* histo) {
    char* d_data;
    unsigned int* d_histo;

    CUDA_CHECK(hipMalloc((void**)&d_data, length * sizeof(char)));
    CUDA_CHECK(hipMemcpy(d_data, data, length * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&d_histo, NUM_BINS * sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_histo, 0, NUM_BINS * sizeof(unsigned int)));

    dim3 dimBlock(1024);
    dim3 dimGrid(cdiv(length, dimBlock.x));

    histo_private_kernel_shared_memory<<<dimGrid, dimBlock>>>(d_data, length, d_histo);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(histo, d_histo, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_histo));
}

void histogram_parallel_with_thread_coarsing(char* data, unsigned int length, unsigned int* histo) {
    char* d_data;
    unsigned int* d_histo;

    CUDA_CHECK(hipMalloc((void**)&d_data, length * sizeof(char)));
    CUDA_CHECK(hipMemcpy(d_data, data, length * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&d_histo, NUM_BINS * sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_histo, 0, NUM_BINS * sizeof(unsigned int)));

    dim3 dimBlock(1024);
    dim3 dimGrid(cdiv(length, dimBlock.x * CFACTOR));  // here we need to launch proportionally less threads

    histo_private_kernel_thread_coarsing<<<dimGrid, dimBlock>>>(d_data, length, d_histo);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(histo, d_histo, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_histo));
}

void histogram_parallel_thread_coarsing_and_coalesced_memory_acess(char* data, unsigned int length,
                                                                   unsigned int* histo) {
    char* d_data;
    unsigned int* d_histo;

    CUDA_CHECK(hipMalloc((void**)&d_data, length * sizeof(char)));
    CUDA_CHECK(hipMemcpy(d_data, data, length * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&d_histo, NUM_BINS * sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_histo, 0, NUM_BINS * sizeof(unsigned int)));

    dim3 dimBlock(1024);
    dim3 dimGrid(getNumSMs() * BLOCKS_PER_SM);  // here we ensure that each SM has multiple units of work to process, so
                                                // we can get some benefit for coarsing

    histo_private_kernel_thread_coarsing_and_coalesced_memory_accesss<<<dimGrid, dimBlock>>>(d_data, length, d_histo);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(histo, d_histo, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_histo));
}

void histogram_sequential(char* data, unsigned int length, unsigned int* histo) {
    for (unsigned int i = 0; i < length; ++i) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            histo[alphabet_position / BIN_SIZE]++;
        }
    }
}

float benchmark_histogram(void (*func)(char*, unsigned int, unsigned int*), char* data, unsigned int length,
                          unsigned int* histo, int warmup = 25, int reps = 100) {
    memset(histo, 0, NUM_BINS * sizeof(unsigned int));

    for (int i = 0; i < warmup; ++i) {
        func(data, length, histo);
        memset(histo, 0, NUM_BINS * sizeof(unsigned int));
    }

    hipEvent_t iterStart, iterStop;
    hipEventCreate(&iterStart);
    hipEventCreate(&iterStop);

    float totalTime_ms = 0.0f;

    for (int i = 0; i < reps; ++i) {
        clear_l2();
        memset(histo, 0, NUM_BINS * sizeof(unsigned int));
        hipEventRecord(iterStart);
        func(data, length, histo);
        hipEventRecord(iterStop);
        hipEventSynchronize(iterStop);

        float iterTime = 0.0f;
        hipEventElapsedTime(&iterTime, iterStart, iterStop);
        totalTime_ms += iterTime;
    }

    hipEventDestroy(iterStart);
    hipEventDestroy(iterStop);

    return totalTime_ms / reps;
}

int main(int argc, char const* argv[]) {
    unsigned int length = 100000000;
    unsigned int histo_parallel[NUM_BINS] = {0};
    unsigned int histo_sequential[NUM_BINS] = {0};
    unsigned int histo_private[NUM_BINS] = {0};
    unsigned int histo_private_with_shared_memory[NUM_BINS] = {0};
    unsigned int histo_private_with_thread_coarsing[NUM_BINS] = {0};
    unsigned int histo_private_with_thread_coarsing_and_coalased_memory_access[NUM_BINS] = {0};

    char* data = generate_random_text(length);
    if (data == NULL) {
        return 1;
    }

    printf("Configuration:\n");
    printf("BIN_SIZE: %d\n", BIN_SIZE);
    printf("NUM_BINS: %d\n\n", NUM_BINS);

    printf("Benchmarking parallel histogram...\n");
    float parallel_time = benchmark_histogram(histogram_parallel, data, length, histo_parallel);

    printf("Benchmarking parallel private histogram...\n");
    float private_time = benchmark_histogram(histogram_parallel_private, data, length, histo_private);

    printf("Benchmarking parallel private histogram with shared memory...\n");
    float private_shared_memory_time = benchmark_histogram(histogram_parallel_private_with_shared_memory, data, length,
                                                           histo_private_with_shared_memory);

    printf("Benchmarking parallel private histogram with thread coarsing...\n");
    float private_thread_coarsing_time =
        benchmark_histogram(histogram_parallel_with_thread_coarsing, data, length, histo_private_with_thread_coarsing);

    printf("Benchmarking parallel private histogram with thread coarsing and coalesced memory access...\n");
    float private_thread_coarsing_and_coalesced_memory_access_time =
        benchmark_histogram(histogram_parallel_thread_coarsing_and_coalesced_memory_acess, data, length,
                            histo_private_with_thread_coarsing_and_coalased_memory_access);

    printf("Benchmarking sequential histogram...\n");
    float sequential_time = benchmark_histogram(histogram_sequential, data, length, histo_sequential, 10, 10);

    printf("\nResults:\n");
    printf("Parallel Implementation:\n");
    printf("Average time: %.3f ms\n", parallel_time);
    // print_histogram_bins(histo_parallel, "Histogram values");

    printf("\nParallel Private Implementation:\n");
    printf("Average time: %.3f ms\n", private_time);
    // print_histogram_bins(histo_private, "Histogram values");

    printf("\nParallel Private with Shared memory Implementation:\n");
    printf("Average time: %.3f ms\n", private_shared_memory_time);
    // print_histogram_bins(histo_private_with_shared_memory, "Histogram values");

    printf("\nParallel Private with thread coarsing Implementation:\n");
    printf("Average time: %.3f ms\n", private_thread_coarsing_time);
    // print_histogram_bins(histo_private_with_thread_coarsing, "Histogram values");

    printf("\nParallel Private with thread coarsing and coalesced memory access Implementation:\n");
    printf("Average time: %.3f ms\n", private_thread_coarsing_and_coalesced_memory_access_time);
    print_histogram_bins(histo_private_with_thread_coarsing_and_coalased_memory_access, "Histogram values");

    printf("\nSequential Implementation:\n");
    printf("Average time: %.3f ms\n", sequential_time);
    // print_histogram_bins(histo_sequential, "Histogram values");

    printf("\nSpeedups:\n");
    printf("Parallel vs Sequential: %.2fx\n", sequential_time / parallel_time);

    printf("Parallel Private vs Sequential: %.2fx\n", sequential_time / private_time);
    printf("Parallel Private vs Parallel: %.2fx\n", parallel_time / private_time);

    printf("Parallel Private with Shared memory vs Sequential: %.2fx\n", sequential_time / private_shared_memory_time);
    printf("Parallel Private with Shared memory vs Parallel: %.2fx\n", parallel_time / private_shared_memory_time);

    printf("Parallel with thread coarsing vs Sequential: %.2fx\n", sequential_time / private_thread_coarsing_time);
    printf("Parallel with thread coarsing vs Parallel: %.2fx\n", parallel_time / private_thread_coarsing_time);
    printf("Parallel with thread coarsing vs Parallel Private: %.2fx\n", private_time / private_thread_coarsing_time);
    printf("Parallel with thread coarsing vs Parallel Private with Shared memory: %.2fx\n",
           private_shared_memory_time / private_thread_coarsing_time);

    printf("Parallel with thread coarsing and coalesced memory access vs Sequential: %.2fx\n",
           sequential_time / private_thread_coarsing_and_coalesced_memory_access_time);
    printf("Parallel with thread coarsing and coalesced memory access vs Parallel: %.2fx\n",
           parallel_time / private_thread_coarsing_and_coalesced_memory_access_time);
    printf("Parallel with thread coarsing and coalesced memory access vs Parallel Private: %.2fx\n",
           private_time / private_thread_coarsing_and_coalesced_memory_access_time);
    printf("Parallel with thread coarsing and coalesced memory access vs Parallel Private with Shared memory: %.2fx\n",
           private_shared_memory_time / private_thread_coarsing_and_coalesced_memory_access_time);
    printf("Parallel with thread coarsing and coalesced memory access vs Parallel with thread coarsing: %.2fx\n",
           private_thread_coarsing_time / private_thread_coarsing_and_coalesced_memory_access_time);

    bool results_match = true;
    for (int i = 0; i < NUM_BINS; i++) {
        bool bin_matches =
            histo_sequential[i] == histo_parallel[i] && histo_sequential[i] == histo_private[i] &&
            histo_private[i] == histo_private_with_shared_memory[i] &&
            histo_private_with_shared_memory[i] == histo_private_with_thread_coarsing[i] &&
            histo_private_with_thread_coarsing[i] == histo_private_with_thread_coarsing_and_coalased_memory_access[i];

        if (!bin_matches) {
            results_match = false;
            break;
        }
    }
    printf("Results match: %s\n", results_match ? "Yes" : "No");

    free(data);
    return 0;
}