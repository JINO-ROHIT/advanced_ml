#include <hip/hip_runtime.h>
#include <stdio.h>

#define BIN_SIZE 4
#define NUM_BINS ((26 + BIN_SIZE - 1) / BIN_SIZE)

// --------------------------------------------------------------------------------------------------------------------------------------------------------
// assertion check

#define CUDA_CHECK(call)                                                                                 \
    do {                                                                                                \
        hipError_t error = call;                                                                       \
        if (error != hipSuccess) {                                                                     \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(error));\
            exit(EXIT_FAILURE);                                                                         \
        }                                                                                               \
    } while (0)

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

// ---------------------------------------------------------------------------------------------------------------------------------------------------------
// kernels

__global__ void histo_kernel(char* data, unsigned int length, unsigned int* histo) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd((unsigned int*)&histo[alphabet_position / BIN_SIZE], 1);
        }
    }
}

// using a private histo block and then merging them
__global__ void histo_private_kernel(char* data, unsigned int length, unsigned int* histo) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&histo[blockIdx.x * NUM_BINS + alphabet_position / BIN_SIZE], 1);
        }
    }

    if (blockIdx.x > 0) {
        __syncthreads();
        for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
            unsigned int binValue = histo[blockIdx.x * NUM_BINS + bin];
            if (binValue > 0) {
                atomicAdd(&histo[bin], binValue);
            }
        }
    }
}

// using shared memory with a private histo block
__global__ void histo_private_kernel_shared_memory(char* data, unsigned int length, unsigned int* histo) {
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&histo_s[alphabet_position / BIN_SIZE], 1);
        }
    }
    __syncthreads();

    // Commit to the global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        unsigned int binValue = histo_s[bin];
        if (binValue > 0) {
            atomicAdd(&histo[bin], binValue);
        }
    }
}

// ---------------------------------------------------------------------------------------------------------------------------------------------------------
// kernel functions

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}

void histogram_parallel(char* data, unsigned int length, unsigned int* histo) {
    char* d_data;
    unsigned int* d_histo;

    CUDA_CHECK(hipMalloc((void**)&d_data, length * sizeof(char)));
    CUDA_CHECK(hipMemcpy(d_data, data, length * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&d_histo, NUM_BINS * sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_histo, 0, NUM_BINS * sizeof(unsigned int)));

    dim3 dimBlock(1024);
    dim3 dimGrid(cdiv(length, dimBlock.x));

    histo_kernel<<<dimGrid, dimBlock>>>(d_data, length, d_histo);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(histo, d_histo, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_histo));
}

void histogram_parallel_private(char* data, unsigned int length, unsigned int* histo) {
    char* d_data;
    unsigned int* d_histo;

    dim3 dimBlock(1024);
    dim3 dimGrid(cdiv(length, dimBlock.x));

    CUDA_CHECK(hipMalloc((void**)&d_data, length * sizeof(char)));
    CUDA_CHECK(hipMemcpy(d_data, data, length * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&d_histo,
                          NUM_BINS * dimGrid.x * sizeof(unsigned int)));  // here we allocate NUM_BINS for every block

    histo_private_kernel<<<dimGrid, dimBlock>>>(d_data, length, d_histo);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(histo, d_histo, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_histo));
}