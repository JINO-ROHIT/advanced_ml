#include <hip/hip_runtime.h>
#include <stdio.h>

#define BIN_SIZE 4
#define NUM_BINS ((26 + BIN_SIZE - 1) / BIN_SIZE)

#define CUDA_CHECK(call)                                                                                 \
    do {                                                                                                \
        hipError_t error = call;                                                                       \
        if (error != hipSuccess) {                                                                     \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(error));\
            exit(EXIT_FAILURE);                                                                         \
        }                                                                                               \
    } while (0)

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

// ---------------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void histo_kernel(char* data, unsigned int length, unsigned int* histo) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd((unsigned int*)&histo[alphabet_position / BIN_SIZE], 1);
        }
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}

void histogram_parallel(char* data, unsigned int length, unsigned int* histo) {
    char* d_data;
    unsigned int* d_histo;

    CUDA_CHECK(hipMalloc((void**)&d_data, length * sizeof(char)));
    CUDA_CHECK(hipMemcpy(d_data, data, length * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&d_histo, NUM_BINS * sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_histo, 0, NUM_BINS * sizeof(unsigned int)));

    dim3 dimBlock(1024);
    dim3 dimGrid(cdiv(length, dimBlock.x));

    histo_kernel<<<dimGrid, dimBlock>>>(d_data, length, d_histo);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(histo, d_histo, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_histo));
}

int main() {
    const char data[] = "abcdefghijklmnopqrstuvwxyz";
    unsigned int histo[NUM_BINS] = {0};
    histogram_parallel((char*)data, sizeof(data) - 1, histo);

    printf("Histogram bins:\n");
    for (int i = 0; i < NUM_BINS; ++i) {
        printf("Bin %d: %u\n", i, histo[i]);
    }
    return 0;
}