#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_runtime.h>

__global__ void blurKernel(unsigned char *in, unsigned char *out, int w, int h, int channels, int BLUR_SIZE) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < w && row < h) {

        for (int c = 0; c < channels; ++c) {
            int pixVal = 0;
            int pixels = 0;

            // Average of surrounding blur_size x blur_size box
            for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
                for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
                    int curRow = row + blurRow;
                    int curCol = col + blurCol;

                    if (curRow >= 0 && curRow < h && curCol >= 0 && curCol < w) {

                        pixVal += in[(curRow * w + curCol) * channels + c];
                        ++pixels;
                    }
                }
            }

            out[(row * w + col) * channels + c] = (unsigned char)(pixVal / pixels);
        }
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}

torch::Tensor gaussian_blur(torch::Tensor img, int blurSize) {
    assert(img.device().type() == torch::kCUDA);
    assert(img.dtype() == torch::kByte);

    const auto height = img.size(0);
    const auto width = img.size(1);
    const auto channels = img.size(2);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(cdiv(width, dimBlock.x), cdiv(height, dimBlock.y));

    auto result = torch::empty({height, width, channels}, 
                              torch::TensorOptions().dtype(torch::kByte).device(img.device()));

    blurKernel<<<dimGrid, dimBlock, 0, torch::cuda::getCurrentCUDAStream()>>>(
        img.data_ptr<unsigned char>(), 
        result.data_ptr<unsigned char>(), 
        width, height, channels, blurSize);

    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return result;
}