#include "hip/hip_runtime.h"
#include <iostream>

#include "stencil.h"

// Global coefficient variables definition
int c0 = 0;
int c1 = 1;
int c2 = 1;
int c3 = 1;
int c4 = 1;
int c5 = 1;
int c6 = 1;


__global__ void stencil_kernel(float* in, float* out, unsigned int N, int c0, int c1, int c2, int c3, int c4, int c5,
                               int c6) {
    unsigned int i = blockIdx.z * blockDim.z + threadIdx.z;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1 && i < N - 1 && j >= 1 && j < N - 1 && k >= 1 && k < N - 1) {
        out[i * N * N + j * N + k] = c0 * in[i * N * N + j * N + k] + 
                                     c1 * in[i * N * N + j * N + (k - 1)] +
                                     c2 * in[i * N * N + j * N + (k + 1)] + 
                                     c3 * in[i * N * N + (j - 1) * N + k] +
                                     c4 * in[i * N * N + (j + 1) * N + k] + 
                                     c5 * in[(i - 1) * N * N + j * N + k] +
                                     c6 * in[(i + 1) * N * N + j * N + k];
    }
}

void stencil_3d_parallel_basic(float* in, float* out, unsigned int N, int c0, int c1, int c2, int c3, int c4, int c5,
                               int c6) {
    float *d_in, *d_out;
    hipError_t error;

    error = hipMalloc((void**)&d_in, N * N * N * sizeof(float));
    if (error != hipSuccess) {
        std::cout << "hipMalloc d_in failed: " << hipGetErrorString(error) << std::endl;
        return;
    }

    error = hipMalloc((void**)&d_out, N * N * N * sizeof(float));
    if (error != hipSuccess) {
        std::cout << "hipMalloc d_out failed: " << hipGetErrorString(error) << std::endl;
        return;
    }

    error = hipMemcpy(d_in, in, N * N * N * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        std::cout << "hipMemcpy to device failed: " << hipGetErrorString(error) << std::endl;
        return;
    }

    dim3 dimBlock(OUT_TILE_DIM_SMALL, OUT_TILE_DIM_SMALL, OUT_TILE_DIM_SMALL);
    dim3 dimGrid(cdiv(N, dimBlock.x), cdiv(N, dimBlock.y), cdiv(N, dimBlock.z));

    stencil_kernel<<<dimGrid, dimBlock>>>(d_in, d_out, N, c0, c1, c2, c3, c4, c5, c6);

    error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "Kernel launch failed: " << hipGetErrorString(error) << std::endl;
    }
    hipDeviceSynchronize();

    error = hipMemcpy(out, d_out, N * N * N * sizeof(float), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        std::cout << "hipMemcpy to host failed: " << hipGetErrorString(error) << std::endl;
    }

    hipFree(d_in);
    hipFree(d_out);
}